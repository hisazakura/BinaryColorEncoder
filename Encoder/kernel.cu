#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__  
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <direct.h> 
#include <algorithm>

#include <chrono>

#include "binaryfile.h"

#define BLOCK_SIZE 512
#define USE_STABILIZER true // used just to artistic measures, will slightly hurt performance

class encoder
{
public:
    struct square {
        int x;
        int y;
        int size;
        square() {
            this->x = 0;
            this->y = 0;
            this->size = 0;
        }
        explicit square(int x, int y, int size) {
            this->x = x;
            this->y = y;
            this->size = size;
        }
    };
    encoder(binaryfile& file);
    encoder::square* encodeFrame(int frame, int* squareCount);

private:
    binaryfile& bin;
    int width, height, frames;
    int* d_integral;
    int* d_convolution;
    int* d_indices;
    int* d_sortedIndices;
    int* d_indicesCount;
    encoder::square* d_squares;
    int* d_squaresCount;
};

bool integralImage(int* integralBuffer, const bool* buffer, int width, int height) {
    if (!integralBuffer || !buffer || width <= 0 || height <= 0) {
        return false;
    }

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            int sum = buffer[y * width + x];

            if (x > 0) {
                sum += integralBuffer[y * width + (x - 1)];
            }
            if (y > 0) {
                sum += integralBuffer[(y - 1) * width + x];
            }
            if (x > 0 && y > 0) {
                sum -= integralBuffer[(y - 1) * width + (x - 1)];
            }

            integralBuffer[y * width + x] = sum;
        }
    }

    return true;
}

// from convolution result
__global__ void reduceIntegralRoi(int* integral, const int width, const int height, const int* indices, const int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // get position from convolution array
    int indice = indices[0];
    int row = indice / (width - size + 1);
    int col = indice % (width - size + 1);

    int roiHeight = height - row;
    int roiWidth = width - col;

    if (idx >= roiWidth * roiHeight) return;

    int roi_i = idx / roiWidth;
    int roi_j = idx % roiWidth;

    int int_i = roi_i + row;
    int int_j = roi_j + col;

    // multiplication square
    if (roi_i < size && roi_j < size) {
        integral[int_i * width + int_j] -= (roi_i + 1) * (roi_j + 1);
    }

    // extend right
    else if (roi_i < size && roi_j < roiWidth) {
        integral[int_i * width + int_j] -= (roi_i + 1) * size;
    }

    // extend down
    else if (roi_i < roiHeight && roi_j < size) {
        integral[int_i * width + int_j] -= size * (roi_j + 1);
    }

    // extend beyond
    else if (roi_i < roiHeight && roi_j < roiWidth) {
        integral[int_i * width + int_j] -= size * size;
    }
}

__global__ void convolveIntegral(int* buffer, const int* integral, const int width, const int height, const int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int totalElements = (width - size + 1) * (height - size + 1);
    if (idx >= totalElements) return;

    int i = idx / (width - size + 1);
    int j = idx % (width - size + 1);

    // guarantee that it wont go out of bound both right and down
    int val = integral[(i + size - 1) * width + (j + size - 1)];
    if (i > 0) val -= integral[(i - 1) * width + (j + size - 1)];
    if (j > 0) val -= integral[(i + size - 1) * width + (j - 1)];
    if (i > 0 && j > 0) val += integral[(i - 1) * width + (j - 1)];

    buffer[i * (width - size + 1) + j] = val;
}

__global__ void findIndices(const int* array, int* indices, int* count, const int target, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    if (array[idx] == target) {
        int pos = atomicAdd(count, 1);
        indices[pos] = idx;
    }
}

// from convolution result
__global__ void addSquare(encoder::square* squares, int* squaresCount, const int* indices, const int width, const int height, const int size, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;
    int indice = indices[idx];
    int row = indice / (width - size + 1);
    int col = indice % (width - size + 1);

    int pos = atomicAdd(squaresCount, 1);
    squares[pos].x = col;
    squares[pos].y = row;
    squares[pos].size = size;
}

void getLaunchConfig(const int n, int* blockSize, int* gridSize) {
    *blockSize = BLOCK_SIZE;
    *gridSize = (n + *blockSize - 1) / *blockSize;
}

encoder::encoder(binaryfile& bin) : bin(bin) {
    width = bin.getMetadata<int>(binaryfile::metadatatype::Width);
    height = bin.getMetadata<int>(binaryfile::metadatatype::Height);
    frames = bin.getMetadata<int>(binaryfile::metadatatype::Frames);

    hipMalloc(&d_integral, width * height * sizeof(int));
    hipMalloc(&d_convolution, width * height * sizeof(int));
    hipMalloc(&d_indices, width * height * sizeof(int));
    hipMalloc(&d_sortedIndices, width * height * sizeof(int));
    hipMalloc(&d_indicesCount, sizeof(int));
    hipMalloc((void**)&d_squares, width * height * sizeof(encoder::square));
    hipMalloc(&d_squaresCount, sizeof(int));

    hipMemset(d_indicesCount, 0, sizeof(int));
}

encoder::square* encoder::encodeFrame(int frame, int* squareCount) {
    bool* h_buffer = new bool[width * height];
    bin.loadFrame(h_buffer, width, height, frame);

    int* h_integral = new int[width * height];
    integralImage(h_integral, h_buffer, width, height);
    hipMemcpy(d_integral, h_integral, width * height * sizeof(int), hipMemcpyHostToDevice);

    delete[] h_buffer;
    delete[] h_integral;

    int h_indicesCount = 0;

    int kernelSize = min(width, height);
    int blockSize, gridSize;

    hipMemset(d_squaresCount, 0, sizeof(int));

    while (kernelSize > 0) {
        int convolutionSize = (width - kernelSize + 1) * (height - kernelSize + 1);

        getLaunchConfig(convolutionSize, &blockSize, &gridSize);
        convolveIntegral << <gridSize, blockSize >> > (d_convolution, d_integral, width, height, kernelSize);
        hipDeviceSynchronize();

        getLaunchConfig(convolutionSize, &blockSize, &gridSize);
        hipMemset(d_indicesCount, 0, sizeof(int));
        findIndices << <gridSize, blockSize >> > (d_convolution, d_indices, d_indicesCount, kernelSize * kernelSize, convolutionSize);
        hipDeviceSynchronize();

        hipMemcpy(&h_indicesCount, d_indicesCount, sizeof(int), hipMemcpyDeviceToHost);

        // trivial condition: add all indices if kernel size is 1
        if (kernelSize == 1) {
            getLaunchConfig(h_indicesCount, &blockSize, &gridSize);
            addSquare << <gridSize, blockSize >> > (d_squares, d_squaresCount, d_indices, width, height, kernelSize, h_indicesCount);
            hipDeviceSynchronize();
            break;
        }

        // sort the indices to stabilize
        if (USE_STABILIZER) {
            int* h_indices = new int[h_indicesCount];
            hipMemcpy(h_indices, d_indices, h_indicesCount * sizeof(int), hipMemcpyDeviceToHost);

            std::sort(h_indices, h_indices + h_indicesCount);

            hipMemcpy(d_indices, h_indices, h_indicesCount * sizeof(int), hipMemcpyHostToDevice);
        }
        
        if (h_indicesCount > 0) {
            addSquare << <1, 1 >> > (d_squares, d_squaresCount, d_indices, width, height, kernelSize, 1);
            hipDeviceSynchronize();

            getLaunchConfig(width * height, &blockSize, &gridSize);
            reduceIntegralRoi << <gridSize, blockSize >> > (d_integral, width, height, d_indices, kernelSize);
            hipDeviceSynchronize();

            continue;
        }

        kernelSize -= 1;
    }

    hipMemcpy(squareCount, d_squaresCount, sizeof(int), hipMemcpyDeviceToHost);

    encoder::square* h_squares = new encoder::square[*squareCount];
    hipMemcpy(h_squares, d_squares, *squareCount * sizeof(encoder::square), hipMemcpyDeviceToHost);

    return h_squares;
}

int main(int argc, char* argv[])
{
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <filepath> <outputpath>\n";
        return 1;
    }

    std::string filePath = argv[1];
    std::string ouputPath = argv[2];
    std::cout << "File path: " << filePath << std::endl;
    std::cout << "Output path: " << ouputPath << std::endl;

    binaryfile bin(filePath);

    int width = bin.getMetadata<int>(binaryfile::metadatatype::Width);
    int height = bin.getMetadata<int>(binaryfile::metadatatype::Height);
    int frameCount = bin.getMetadata<int>(binaryfile::metadatatype::Frames);
    float fps = bin.getMetadata<float>(binaryfile::metadatatype::Fps);

    std::cout << "Width: " << width << ", Height: " << height << ", FPS: " << fps << ", Frames: " << frameCount << std::endl;

    if (height > 1024) {
        fprintf(stderr, "Height more than 1024 pixels is not yet supported.");
        return 1;
    }

    std::ofstream outputFile(ouputPath, std::ios::out | std::ios::binary);
    if (!outputFile) {
        std::cerr << "Error: Could not create or open file!" << std::endl;
        return 1;
    }
    if (!outputFile.is_open()) {
        std::cerr << "Error opening output file!" << std::endl;
        return 1;
    }

    int* squaresPerFrame = new int[frameCount];
    std::vector<encoder::square> squares = std::vector<encoder::square>();

    std::cout << "Loading data..." << std::endl;
    auto loadStart = std::chrono::high_resolution_clock::now();
    encoder encoder = encoder::encoder(bin);
    auto loadEnd = std::chrono::high_resolution_clock::now();
    std::cout << "Data load took " << std::chrono::duration_cast<std::chrono::milliseconds>(loadEnd - loadStart).count() << "ms." << std::endl;

    auto encodeStart = std::chrono::high_resolution_clock::now();
    for (int frameIndex = 0; frameIndex < frameCount; frameIndex++) {
        std::cout << "\rEncoding frame " << frameIndex + 1 << " out of " << frameCount << "...";
        int squareCount = 0;
        encoder::square* frameSquares = encoder.encodeFrame(frameIndex, &squareCount);

        squaresPerFrame[frameIndex] = squareCount;
        squares.insert(squares.end(), frameSquares, frameSquares + squareCount);

        delete[] frameSquares;
    }
    auto encodeEnd = std::chrono::high_resolution_clock::now();

    std::cout << std::endl << "Encoder took " << std::chrono::duration_cast<std::chrono::milliseconds>(encodeEnd - encodeStart).count() << "ms and resulted in " << squares.size() << " squares!" << std::endl;
    
    std::cout << "Writing to file..." << std::endl;

    auto writeStart = std::chrono::high_resolution_clock::now();
    int startingIndex = 0;
    for (int frameIndex = 0; frameIndex < frameCount; frameIndex++) {
        uint32_t frameSquareCount = static_cast<uint32_t>(squaresPerFrame[frameIndex]);
        outputFile.write(reinterpret_cast<char*>(&frameSquareCount), sizeof(uint32_t));
        for (int squareIndex = 0; squareIndex < squaresPerFrame[frameIndex]; squareIndex++) {
            encoder::square square = squares[startingIndex + squareIndex];

            uint16_t x = static_cast<uint16_t>(square.x);
            uint16_t y = static_cast<uint16_t>(square.y);
            uint16_t size = static_cast<uint16_t>(square.size);

            outputFile.write(reinterpret_cast<char*>(&y), sizeof(uint16_t));
            outputFile.write(reinterpret_cast<char*>(&x), sizeof(uint16_t));
            outputFile.write(reinterpret_cast<char*>(&size), sizeof(uint16_t));
        }
        startingIndex += squaresPerFrame[frameIndex];
    }
    auto writeEnd = std::chrono::high_resolution_clock::now();

    std::cout << "Writing to file took " << std::chrono::duration_cast<std::chrono::milliseconds>(writeEnd - writeStart).count() << "ms." << std::endl;

    return 0;
}